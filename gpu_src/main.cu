#include "hip/hip_runtime.h"
//#include "simple_la.cuh"
#include <thrust/complex.h>
#include <stdio.h>
#include <iostream>

using namespace std;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}


// copy matrix/vector 
__global__ void matrixCopy(thrust::complex<double> *src, thrust::complex<double>* dst, int N){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int idy = blockIdx.y * blockDim.y + threadIdx.y;
        dst[idx*N + idy] = src[idx*N + idy];
}

// matrix multiplcation
__global__ void matrixMul(thrust::complex<double>* srcA, 
			thrust::complex<double>* srcB, 
			thrust::complex<double>* dstC,
			int N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;

	thrust::complex<double>  c_temp = 0.0f;
	if (idx < N && idy < N){
		for(int i = 0; i < N; ++i){
			c_temp += srcA[idy*N + i] * srcB[i*N + idx];// + 0.0f;
		}
		dstC[idy * N + idx] += c_temp;
	}
	__syncthreads();

}

// conjugate tranpose
__global__ void conj_transpose(thrust::complex<double>* src, thrust::complex<double>* dst){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;

	dst[idy*blockDim.y+idx] = thrust::conj(src[idx*blockDim.x+idy]);
}
	
void cov_matrix(thrust::complex<double>* src, int M, int N, thrust::complex<double>* &dst){
	size_t size = (M*N)*sizeof(thrust::complex<double>);
	thrust::complex<double>* srcT = (thrust::complex<double> *)malloc(size);
	if(M*N <= 32*32)
		dim3 grid_block(1,1);
	else if(M*N/2 <= 32*32)
		dim3 grid_block(2,2);
		
	dim3 block_thread(N/grid_block.x, N/grid_block.y);


	checkCuda(hipHostFree(srcT));
}

int main(void){
	int N = 2;
	size_t size = (N*N)*sizeof(thrust::complex<double>);

	thrust::complex<double>* h_A = (thrust::complex<double> *)malloc(size);
	thrust::complex<double>* h_C = (thrust::complex<double> *)malloc(size);
	for(int i = 0; i < N*N; ++i){
	//	h_A[i] = thrust::complex<double>(rand()/(float)RAND_MAX, rand()/(float)RAND_MAX);
		h_A[i] = thrust::complex<double>(i, 0);
	}

	thrust::complex<double>* d_A = NULL;
	thrust::complex<double>* d_C = NULL;

	checkCuda(hipMalloc((void**)&d_A, size));
	checkCuda(hipMalloc((void**)&d_C, size));

	checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

	dim3 grid_block(1,1);
	dim3 block_thread(N/grid_block.x, N/grid_block.y);
	matrixCopy<<<grid_block, block_thread>>>(d_A, d_C, N);

	checkCuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

	for(int i = 0; i < N*N; ++i){
		if(h_C[i] != h_A[i]) 
			printf("FUCK\n");
	}
	printf("DONE\n");

	matrixMul<<<grid_block, block_thread>>>(d_A,d_A, d_C, N);
	checkCuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

	for(int i=0;i<N;++i){
		for(int j=0;j<N;++j){
			cout << h_C[i*N+j] << " ";
		}
		cout << endl;
	}

	conj_transpose<<<grid_block, block_thread>>>(d_A, d_C);
	checkCuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
	for(int i=0;i<N;++i){
		for(int j=0;j<N;++j){
			cout << h_C[i*N+j] << " ";
		}
		cout << endl;
	}
			

	checkCuda(hipHostFree(h_A));
	checkCuda(hipHostFree(h_C));
	checkCuda(hipFree(d_A));
	checkCuda(hipFree(d_C));
}

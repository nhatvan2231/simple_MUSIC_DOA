#include "hip/hip_runtime.h"
#include "simple_la.cuh"


// copy matrix/vector 
__global__ void matrix_copy(thrust::complex<double> *src, thrust::complex<double>* dst, int M){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	dst[idx*M + idy] = src[idx*M + idy];
}

//// calculate matrix multiplication
//int matrix_mul(std::complex<double> *srcA,
//							std::complex<double> *srcB, 
//							int M_A, int N_A, 
//							int M_B, int N_B,
//							std::complex<double>* &dst){
//	if(N_A != M_B){
//		printf("Invalid matrix multiplication\n");
//		return 1;
//	}
//	for(int i = 0; i < M_A; ++i){
//		for(int j = 0; j < N_B; ++j){
//			for(int k = 0; k < N_A; ++k){
//				dst[i*N_B+j] += srcA[i*N_A+k] * srcB[k*N_B+j];
//			}
//		}
//	}
//	return 0;
//}
//
//// calculate the conjugate transpose of a matrix or vector
//int conj_transpose(std::complex<double> *src, int M, int N, std::complex<double>* &dst){
//	for(int i=0; i<N; ++i){
//		for(int j=0; j<M; ++j){
//			dst[i*M+j] = conj(src[j*N+i]);
//		}
//	}
//	return 0;
//}
//
//// calculate the covariance matrix without normalize
//int cov_matrix(std::complex<double> *src, int M, int N, std::complex<double>* &dst){
//	std::complex<double> *srcT = (std::complex<double>*)malloc(sizeof(std::complex<double>)*N*M);
//	conj_transpose(src, M, N, srcT);
//	matrix_mul(src, srcT, M, N, N, M, dst);
//	free(srcT);
//	return 0;
//}
//
//// get the i column of the matrix
//int get_col(std::complex<double>* src, int M, int N, int col_i, std::complex<double>* &dst){
//	for(int i=0; i<M; ++i){
//		dst[i] = src[i*N + col_i];
//	}
//	return 0;
//}
//
//// get the identity matrix with size x size
//int identity_mat(int size, std::complex<double>* &dst){
//	for(int i=0; i<size; ++i){
//		for(int j=0; j<size; ++j){
//			dst[i*size +j] = std::complex<double>(0,0);
//			if(i==j) dst[i*size +j] = std::complex<double>(1,0);
//		}
//	}
//	return 0;
//}
//
//// calculate the norm of vector
//double vec_norm(std::complex<double> *src, int size){
//	double result=0;
//	for(int i=0; i<size; ++i){
//		result += norm(src[i]);
//	}
//	return sqrt(result);
//}
//
//// generate random number
//double random_num(double min=1.0, double max=10.0){
//	std::random_device rd;
//	std::mt19937 gen(rd());
//	std::uniform_real_distribution<double> dis(min, max);
//	return dis(gen);
//}
